#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <chrono>

__global__ void function(float* dA, float* dB, float* dC, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	float ab, sum = 0.f;
	if (i < size)
	{
		ab = dA[i] * dB[i];
		for (j = 0; j < 100; j++) sum = sum + sinf(j + ab);
		dC[i] = sum;
	}
}

int main() {
    float timerValueGPU, timerValueCPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int nStream = 1; // ����� CUDA-�������
    int size = 512 * 50000 / nStream; // ������ ������� �������
    int N_thread = 512; // ����� ����� � �����
    int N_blocks, i;
    // ��������� ������ ��� �������� hA,hB,hC ��� host
    unsigned int mem_size = sizeof(float) * size;
    const int num_streams = 1;

    hipStream_t streams[num_streams];

    float* hA, * hB, * hC;
    float* dA, * dB, * dC;

    // ��������� pinned-������
    hipHostAlloc((void**)&hA, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hB, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hC, mem_size, hipHostMallocDefault);

    // ������������� ������
    for (i = 0; i < size; i++)
    {
        hA[i] = sinf(i); hB[i] = cosf(2.0f * i - 5.0f); hC[i] = 0.0f;
    }

    hipMalloc((void**)&dA, mem_size);
    hipMalloc((void**)&dB, mem_size);
    hipMalloc((void**)&dC, mem_size);

    // �������� �������
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
    }
        
    hipEventRecord(start, 0);
    // ����������� ��������
    int chunk_size = size / num_streams;
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        hipMemcpyAsync(dA + offset, hA + offset, chunk_size * sizeof(float),
            hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(dB + offset, hB + offset, chunk_size * sizeof(float),
            hipMemcpyHostToDevice, streams[i]);
    }

    int threadsPerBlock = 512;
    int blocksPerGrid = (chunk_size + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        function <<< blocksPerGrid, threadsPerBlock, 0, streams[i] >>> (dA + offset, dB + offset, dC + offset, chunk_size);
    }

    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        hipMemcpyAsync(hC + offset, dC + offset, chunk_size * sizeof(float),
            hipMemcpyDeviceToHost, streams[i]);
    }
    // ���������� GPU ��������
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueGPU, start, stop);
    printf("\n GPU calculation time: %f ms\n", timerValueGPU);

    auto start_cpu = std::chrono::high_resolution_clock::now();
    for (i = 0; i < size; i++) {
        int j;
        float ab, sum = 0.f;
        ab = hA[i] * hB[i];
        for (j = 0; j < 100; j++) sum = sum + sinf(j + ab);
        hC[i] = sum;
    }
    auto end_cpu = std::chrono::high_resolution_clock::now();
    timerValueCPU = std::chrono::duration<float, std::milli>(end_cpu - start_cpu).count();
    printf("\n CPU calculation time: %f ms\n", timerValueCPU);
    // ����� ������������ ���������
    printf("\n Rate: %f x\n", timerValueCPU / timerValueGPU);

    // �������������
    for (int i = 0; i < num_streams; i++)
        hipStreamSynchronize(streams[i]);

    // ������������ ��������
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }

    return 0;
}