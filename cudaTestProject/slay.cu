#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <chrono>

double EPS = 1.e-15;
int N = 10240;

__global__ void Solve(double* dA, double* dF, double* dX0, double* dX1, int size) {
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	if (t < size) {
		double sum = 0.0;
		double aa = dA[t + t * size]; // ��� ����������������� �������
		for (int j = 0; j < size; j++) {
			// sum += dA[t + j * size] * dX0[j]; // ��� ������� �������
			sum += dA[j + t * size] * dX0[j];    // ��� ����������������� �������
		}
		dX1[t] = dX0[t] + (dF[t] - sum) / aa;
	}
}

__global__ void Eps(double* dX0, double* dX1, double* delta, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		delta[i] = fabs(dX1[i] - dX0[i]);
		dX0[i] = dX1[i]; // ���������� X0 ��� ��������� ��������
	}
}

int main() {
	float timerValueGPU, timerValueCPU;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	double* hA, * hF, * hX0, * hX, * hX1, * hDelta;
	double* dA, * dF, * dX0, * dX1, * delta;

	int size = N * N; // ������ ������� �������
	int N_thread = 512; // ����� ����� � �����
	int N_blocks = (N + N_thread - 1) / N_thread;
	unsigned int mem_sizeA = sizeof(double) * size; // ������ ��� �������
	unsigned int mem_sizeX = sizeof(double) * N; // ������ ��� ��������

	// ��������� ������ �� host
	hA = (double*)malloc(mem_sizeA); // ������� �
	hF = (double*)malloc(mem_sizeX); // ������ ����� ������� F
	hX = (double*)malloc(mem_sizeX); // ������ �������
	hX0 = (double*)malloc(mem_sizeX); // ������������ ������� X(n)
	hX1 = (double*)malloc(mem_sizeX); // ������������ ������� X(n+1)
	hDelta = (double*)malloc(mem_sizeX); // ������� |X(n+1)- X(n)|

	// ��������� ������ �� device
	hipMalloc((void**)&dA, mem_sizeA); // ������� A
	hipMalloc((void**)&dF, mem_sizeX); // ������ ����� F
	hipMalloc((void**)&dX0, mem_sizeX); // ������� X(n)
	hipMalloc((void**)&dX1, mem_sizeX); // ������� X(n+1)
	hipMalloc((void**)&delta, mem_sizeX); // ������� |X(n+1)- X(n)|

	// ----------------------GPU ������� -------------------
	hipMalloc((void**)&dA, mem_sizeA);
	hipMalloc((void**)&dF, mem_sizeX);
	hipMalloc((void**)&dX0, mem_sizeX);
	hipMalloc((void**)&dX1, mem_sizeX);
	hipMalloc((void**)&delta, mem_sizeX);

	// ����������� ������ c host �� device
	hipMemcpy(dA, hA, mem_sizeA, hipMemcpyHostToDevice); // ������� A
	hipMemcpy(dF, hF, mem_sizeX, hipMemcpyHostToDevice); // ������ ����� F
	hipMemcpy(dX0, hX0, mem_sizeX, hipMemcpyHostToDevice); // ���������
	// �����������

	// ���������� �������� ������
	for (int i = 0; i < N; i++) {
		hF[i] = 1.0;
		hX0[i] = 0.0;
		for (int j = 0; j < N; j++) {
			//hA[j + i * N] = (i == j) ? 2.0 : 0.1; // ������ ������� 
			hA[i + j * N] = (i == j) ? 2.0 : 0.1; // ������ �����������������
		}
	}

	// GPU Implementation
	{
		hipEventRecord(start, 0);
		hipMemcpy(dA, hA, mem_sizeA, hipMemcpyHostToDevice);
		hipMemcpy(dF, hF, mem_sizeX, hipMemcpyHostToDevice);
		hipMemcpy(dX0, hX0, mem_sizeX, hipMemcpyHostToDevice);

		
		double eps = 1.0;
		int k = 0;
		while (eps > EPS) {
			k++;
			Solve << <N_blocks, N_thread >> > (dA, dF, dX0, dX1, N);
			Eps << <N_blocks, N_thread >> > (dX0, dX1, delta, N);

			hipDeviceSynchronize(); // ������ �������������!

			hipMemcpy(hDelta, delta, mem_sizeX, hipMemcpyDeviceToHost);
			eps = 0.0;
			for (int j = 0; j < N; j++) eps += hDelta[j];
			eps /= N;
		}

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&timerValueGPU, start, stop);
	}

	// CPU Implementation
	{
		auto start_cpu = std::chrono::high_resolution_clock::now();

		double* temp;
		double eps = 1.0;
		int k = 0;
		while (eps > EPS) {
			k++;
			eps = 0.0;

			for (int i = 0; i < N; i++) {
				double sum = 0.0;
				// ��� ������� �������: j + i*N
				// ��� �����������������: i + j*N
				for (int j = 0; j < N; j++)
					sum += hA[i + j * N] * hX0[j];

				hX1[i] = hX0[i] + (hF[i] - sum) / hA[i + i * N];
			}
			eps /= N;

			// ����� ���������� ������ ����������� ������
			temp = hX0;
			hX0 = hX1;
			hX1 = temp;
		}

		auto end_cpu = std::chrono::high_resolution_clock::now();
		timerValueCPU = std::chrono::duration<float, std::milli>(end_cpu - start_cpu).count();
	}
	printf("\n GPU calculation time: %f ms\n", timerValueGPU);
	printf("\n CPU calculation time: %f ms\n", timerValueCPU);
	// ����� ������������ ���������
	printf("\n Rate: %f x\n", timerValueCPU / timerValueGPU);

	// ������������ ������
	free(hA);
	free(hF);
	free(hX0);
	free(hX1);
	free(hDelta);
	hipFree(dA);
	hipFree(dF);
	hipFree(dX0);
	hipFree(dX1);
	hipFree(delta);

	return 0;
}