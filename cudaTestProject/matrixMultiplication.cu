#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <chrono>

#define BLOCK_SIZE 32
__global__ void kernel_global(double* a, double* b, int n, double* c)
{
	int bx = blockIdx.x; // ����� ����� �� x
	int by = blockIdx.y; // ����� ����� �� y
	int tx = threadIdx.x; // ����� ���� � ����� �� x
	int ty = threadIdx.y; // ����� ���� � ����� �� y
	float sum = 0.0f;
	int ia = n * (BLOCK_SIZE * by + ty); // ����� ������ �� A�
	int ib = BLOCK_SIZE * bx + tx; // ����� ������� �� B�
	int ic = ia + ib; // ����� �������� �� ђ
	// ���������� �������� ������� C
	for (int k = 0; k < n; k++) sum += a[ia + k] * b[ib + k * n];
	c[ic] = sum;
}

__global__ void kernel_smem(double* a, double* b, int n, double* c)
{
	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;
	int aBegin = n * BLOCK_SIZE * by, aEnd = aBegin + n - 1;
	int bBegin = BLOCK_SIZE * bx, aStep = BLOCK_SIZE, bStep = BLOCK_SIZE * n;
	float sum = 0.0f;
	__shared__ float as[BLOCK_SIZE][BLOCK_SIZE+1];
	__shared__ float bs[BLOCK_SIZE][BLOCK_SIZE+1];
	for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
	{
		// SMEM - 1
		as[tx][ty] = a[ia + n * ty + tx]; 
		bs[tx][ty] = b[ib + n * ty + tx];
		// SMEM - 3
		as[ty][tx] = a[ia + n * ty + tx];
		bs[ty][tx] = b[ib + n * ty + tx];
		//__syncthreads();
		for (int k = 0; k < BLOCK_SIZE; k++) 
			// SMEM - 1
			// sum += as[k][ty] * bs[tx][k];
			// SMEM - 3
			sum += as [ty][k] * bs [k][tx];
		//__syncthreads();
	}
	c[aBegin + bBegin + ty * n + tx] = sum;
}

int main()
{
	int N = 2048;
	int m, n, k;
	// �������� ����������-�������
	float timerValueGPU, timerValueCPU;
	hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	int numBytes = N * N * sizeof(double);
	double* adev, * bdev, * cdev, * a, * b, * c, * cc, * bT;
	// ��������� ������ �� host
	a = (double*)malloc(numBytes); //������� A
	b = (double*)malloc(numBytes); //������� B
	bT = (double*)malloc(numBytes); //����������������� ������� B
	c = (double*)malloc(numBytes); //������� � ��� GPU-��������
	cc = (double*)malloc(numBytes); //������� � ��� CPU-��������
	// ������� ������� A, B � ����������������� ������� B
	for (n = 0; n < N; n++)
	{
		for (m = 0; m < N; m++)
		{
			a[m + n * N] = 2.0f * m + n; b[m + n * N] = m - n; bT[m + n * N] = n - m;
		}
	}
	// ������� ����� ����� � ������
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);
	// ��������� ������ �� GPU
	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);
	// ---------------- GPU-������� ------------------------
	// ����������� ������ A � B � host �� device
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);
	// ������ �������
	hipEventRecord(start, 0);
	// ������ �������-����
	// !!!!!!! kernel_global <<< blocks, threads >> > (adev, bdev, N, cdev);
	kernel_smem << < blocks, threads >> > (adev, bdev, N, cdev);
	// ������ ������� ���������� GPU-��������
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time %f msec\n", timerValueGPU);
	// �����������, ����������� ������� C � device �� host
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

	// -------------------- CPU-������� --------------------
	// ����� �������
	auto start_cpu = std::chrono::high_resolution_clock::now();
	// ���������� ������� C
	for (n = 0; n < N; n++)
	{
		for (m = 0; m < N; m++)
		{
			cc[m + n * N] = 0.f;
			for (k = 0; k < N; k++) cc[m + n * N] += a[k + n * N] * bT[k + m * N]; // bT !!!
		}
	}
	// ������ ������� ���������� CPU-��������
	auto end_cpu = std::chrono::high_resolution_clock::now();
	timerValueCPU = std::chrono::duration<float, std::milli>(end_cpu - start_cpu).count();
	printf("\n CPU calculation time: %f ms\n", timerValueCPU);
	printf("\n Rate %f x\n", timerValueCPU / timerValueGPU);
	// ������������ ������ �� GPU � CPU
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);
	free(a);
	free(b);
	free(bT);
	free(c);
	free(cc);
	// ����������� ����������-�������
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}