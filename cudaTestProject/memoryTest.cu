#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <chrono>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void function(float* dA, float* dB, float* dC, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) dC[i] = dA[i] + dB[i];
}

int main(int argc, char* argv[])
{// инициализация переменных-событий для таймера
    float timerValueGPU, timerValueCPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float* hA, * hB, * hC, * dA, * dB, * dC;
    int size = 512 * 50000; // размер каждого массива
    int N_thread = 512; // число нитей в блоке
    int N_blocks, i;
    // задание массивов hA, hB, hC для host
    unsigned int mem_size = sizeof(float) * size;
    hipHostAlloc((void**)&hA, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hB, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hC, mem_size, hipHostMallocDefault);
    // выделение памяти на device под массивы hA, hB, hC
    hipMalloc((void**)&dA, mem_size);
    hipMalloc((void**)&dB, mem_size);
    hipMalloc((void**)&dC, mem_size);
    // заполнение массивов hA,hB и обнуление hC
    for (i = 0; i < size; i++)
    {
        hA[i] = 1.0f / ((i + 1.0f) * (i + 1.0f));
        hB[i] = expf(1.0f / (i + 1.0f));
        hC[i] = 0.0f;
    }
    // определение числа блоков
    if ((size % N_thread) == 0)
    {
        N_blocks = size / N_thread;
    }
    else
    {
        N_blocks = (int)(size / N_thread) + 1;
    }
    dim3 blocks(N_blocks);

    // ----------------------GPU вариант -------------------
    // Старт таймера
    //hipEventRecord(start, 0);
    // Копирование массивов с host на device
    hipMemcpy(dA, hA, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, mem_size, hipMemcpyHostToDevice);
    // Запуск функции-ядра
    hipEventRecord(start, 0);
    function <<< N_blocks, N_thread >>> (dA, dB, dC, size);
    // Копирование результат с device на host
    hipMemcpy(hC, dC, mem_size, hipMemcpyDeviceToHost);
    // Остановка таймера и вывод времени
    // вычисления GPU варианта
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueGPU, start, stop);
    printf("\n GPU calculation time: %f ms\n", timerValueGPU);

    // --------------------- CPU вариант --------------------
    // Старт таймера
    auto start_cpu = std::chrono::high_resolution_clock::now();
    // hipEventRecord(start, 0);
    // вычисления
    for (i = 0; i < size; i++) hC[i] = hA[i] + hB[i];
    // Остановка таймера и вывод времени
    // вычисления СPU варианта
    //hipEventRecord(stop, 0);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&timerValueCPU, start, stop);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    timerValueCPU = std::chrono::duration<float, std::milli>(end_cpu - start_cpu).count();
    printf("\n CPU calculation time: %f ms\n", timerValueCPU);
    // Вывод коэффициента ускорения
    printf("\n Rate: %f x\n", timerValueCPU / timerValueGPU);
    // Освобождение памяти на host и device
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    // уничтожение переменных-событий
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
